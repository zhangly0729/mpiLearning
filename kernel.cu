#include "hip/hip_runtime.h"
#include"iostream"
#include"hip/hip_runtime_api.h"
#include""
#include"hipfft/hipfft.h"
using namespace std;
//FFT���任�����ڹ淶���ĺ���
__global__ void normalizing(hipfftComplex* data, int data_len)
{
	int idx = blockDim.x*blockIdx.x + threadIdx.x;
	data[idx].x /= data_len;
	data[idx].y /= data_len;
}
#define Check(call) {		\
	if (call != hipSuccess) \
	{\
		cout << "�к�:" << __LINE__ << endl;\
		cout << "����:" << hipGetErrorString(call) << endl;\
	}\
}
int main()
{
	hipSetDevice(1);
	//uint64_t
	uint64_t Nt =1024LL*1024*200;
	uint64_t datasize =1024LL * 1024 * 200*8;
	const int BATCH = 1;
	//BATCH������������һ��һά���ݣ���BATCH=2ʱ
	//��0-1024��1024-2048��Ϊ����һά�ź���FFT����任
	hipfftComplex* host_in, *host_out, *device_in, *device_out;
	//�����ڴ����뼰��ʼ��--������ҳ�ڴ�
	Check(hipHostMalloc((void**)&host_in, datasize ));
	Check(hipHostMalloc((void**)&host_out, datasize));
//	host_in=(hipfftComplex*)malloc(datasize);
//	host_out=(hipfftComplex*)malloc(datasize);
	//Nt = Nt / 8;
	for (int i = 0; i < Nt; i++)
	{
		host_in[i].x = i + 1;
		host_in[i].y = i + 1;
	}
	//�豸�ڴ�����
	size_t freeMem, totalMem;
	hipMemGetInfo(&freeMem, &totalMem);
	Check(hipMalloc((void**)&device_in, Nt * sizeof(hipfftComplex)));
	hipMemGetInfo(&freeMem, &totalMem);
	//Check(hipMalloc((void**)&device_out, Nt * sizeof(hipfftComplex)));
//	hipMemGetInfo(&freeMem, &totalMem);
	//���ݴ���--H2D
	Check(hipMemcpy(device_in, host_in, Nt * sizeof(hipfftComplex), hipMemcpyHostToDevice));


	//����cufft���
	hipfftHandle cufftForwrdHandle, cufftInverseHandle;
	hipfftResult_t cufftstate;
	cufftstate =hipfftPlan1d(&cufftForwrdHandle, Nt, HIPFFT_C2C, BATCH);
	if (cufftstate)	cout << "cufft plan create failed!" << endl;
	cufftstate=hipfftPlan1d(&cufftInverseHandle, Nt, HIPFFT_C2C, BATCH);

	//ִ��fft���任
	hipfftExecC2C(cufftForwrdHandle, device_in, device_in, HIPFFT_FORWARD);
 
	//���ݴ���--D2H
	Check(hipMemcpy(host_in, device_in, Nt * sizeof(hipfftComplex), hipMemcpyDeviceToHost));
 
	//�����������--���任������
	cout << "���任���:" << endl;
	cout.setf(20);
	for (int i = 0; i < Nt; i++)
	{
		//cout << host_out[i].x << "+j*" << host_out[i].y << endl;
	}
 
	//ִ��fft���任
	hipfftExecC2C(cufftInverseHandle, device_in, device_in, HIPFFT_BACKWARD);
 
	//IFFT�������ֵ��N�������Ҫ��/N����
	dim3 grid(Nt / 128);
	dim3 block(128);
	normalizing << <grid, block >> > (device_in, Nt);
 
	//���ݴ���--D2H
	Check(hipMemcpy(host_in, device_in, Nt * sizeof(hipfftComplex), hipMemcpyDeviceToHost));
 //
	////�����������--���任������
	//cout << "���任���:" << endl;
	//cout.setf(20);
	//for (int i = 0; i < Nt; i++)
	//{
	//	//cout << host_in[i].x << "+j*" << host_in[i].y << endl;
	//}
	////cin.get();
	return 0;
}
