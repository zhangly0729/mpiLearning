#include "hip/hip_runtime.h"
#include"lib.h"
#include <iostream>
#include <complex>
#include <cmath>
#include<stdio.h>

/*
	���cuda������
*/
#define Check(CALL) \
{\
	if (CALL != hipSuccess)\
	{\
		cout << "�к�:" << __LINE__ << endl;\
		cout << "����:" << hipGetErrorString(CALL) << endl;\
	}\
}

/********************************************
****************�������˺�������*************
*********************************************/

/*
	���ڷ�FFT�任�����ݵĹ�һ��
*/
__global__ void normalizing2d(hipfftComplex* data, int data_len, int value)
{
	unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x;
	for (int i = idx* blockDim.x; i < (idx + 1)* blockDim.x; i++)
	{
		if (i < data_len)
		{
			data[i].x /= value;
			data[i].y /= value;
		}
	}

}
__global__ void normalizing3d(hipfftComplex* data, int data_len, int value)
{
	unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x;
	for (int i = idx* blockDim.x; i < (idx + 1)* blockDim.x; i++)
	{
		if (i < data_len)
		{
			data[i].x /= value;
			data[i].y /= value;
		}
	}

}
__global__ void normalizing3d(float* data, int data_len, int value)
{
	unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x;
	for (int i = idx* blockDim.x; i < (idx + 1)* blockDim.x; i++)
	{
		if (i < data_len)
		{
			data[i] /= value;
		}
	}

}
/*
	������-��ȡ��ЧƵ������
*/
__global__ void HAMMING_Window2d(hipfftComplex* data, hipfftComplex* data0, int Nw1, int Nw2, int Nw3, int Nw4, int Nw, int LT, int Nx)
{
	unsigned int ix = threadIdx.x + blockDim.x*blockIdx.x;

	unsigned int idx = LT*ix;

	unsigned int idw = Nw*ix;

	float Hammingw = 0;
	if (ix < Nx)
	{
		for (int iw = 0; iw < LT; iw++)
		{
			if (iw >= Nw1&&iw <= Nw2)
			{
				Hammingw = 0.54f + 0.46f*cos(PI*(iw - Nw1) / (Nw2 - Nw1) - PI);
				data[idx + iw].x = data[idx + iw].x * Hammingw;
				data[idx + iw].y = data[idx + iw].y * Hammingw;
			}
			else if (iw >= Nw3&&iw <= Nw4)
			{
				Hammingw = 0.54f + 0.46f*cos(PI*(Nw3 - iw) / (Nw4 - Nw3) - PI);
				data[idx + iw].x = data[idx + iw].x * Hammingw;
				data[idx + iw].y = data[idx + iw].y * Hammingw;
			}
			else if (iw<Nw1 || iw>Nw4)
			{
				data[idx + iw].x = 0;
				data[idx + iw].y = 0;
			}
		}

		for (int iw = Nw1; iw < Nw4 + 1; iw++)
		{
			data0[idw + iw - Nw1] = data[idx + iw];
		}
	}
}
__global__ void HAMMING_Window3d(hipfftComplex* data, hipfftComplex* data0, int Nw1, int Nw2, int Nw3, int Nw4, int Nw, int LT, int Nx, int Ny)
{
	unsigned int ix = threadIdx.x + blockDim.x*blockIdx.x;
	unsigned int iy = threadIdx.y + blockDim.y*blockIdx.y;
	unsigned int idx = LT*(iy*Ny + ix);
	unsigned int idw = Nw*(iy*Ny + ix);
	float Hammingw = 0;
	if (ix < Ny&&iy < Nx)
	{
		for (int iw = 0; iw < LT; iw++)
		{
			if (iw >= Nw1&&iw <= Nw2)
			{
				Hammingw = 0.54f + 0.46f*cos(PI*(iw - Nw1) / (Nw2 - Nw1) - PI);
				data[idx + iw].x = data[idx + iw].x * Hammingw;
				data[idx + iw].y = data[idx + iw].y * Hammingw;
			}
			else if (iw >= Nw3&&iw <= Nw4)
			{
				Hammingw = 0.54f + 0.46f*cos(PI*(Nw3 - iw) / (Nw4 - Nw3) - PI);
				data[idx + iw].x = data[idx + iw].x * Hammingw;
				data[idx + iw].y = data[idx + iw].y * Hammingw;
			}
			else if (iw<Nw1 || iw>Nw4)
			{
				data[idx + iw].x = 0;
				data[idx + iw].y = 0;
			}
		}

		for (int iw = Nw1; iw < Nw4 + 1; iw++)
		{
			data0[idw + iw - Nw1] = data[idx + iw];
		}
	}
}


/*
	����ЧƵ�����ݻ�ԭ
*/
__global__ void HAMMING_Window_Inverse2d(hipfftComplex* data, hipfftComplex* data0, int Nw1, int Nw2, int Nw3, int Nw4, int Nw, int LT, int Nx)
{
	unsigned int ix = threadIdx.x + blockDim.x*blockIdx.x;
	unsigned int idx = LT*ix;
	unsigned int idw = Nw*ix;
	if (ix < Nx)
	{
		for (int iw = Nw1; iw < Nw4 + 1; iw++)
		{
			data[idx + iw] = data0[idw + iw - Nw1];
		}
	}
}
__global__ void HAMMING_Window_Inverse3d(hipfftComplex* data, hipfftComplex* data0, int Nw1, int Nw2, int Nw3, int Nw4, int Nw, int LT, int Nx, int Ny)
{
	unsigned int ix = threadIdx.x + blockDim.x*blockIdx.x;
	unsigned int iy = threadIdx.y + blockDim.y*blockIdx.y;
	unsigned int idx = LT*(iy*Ny + ix);
	unsigned int idw = Nw*(iy*Ny + ix);
	if (iy < Nx&&ix < Ny)
	{
		for (int iw = Nw1; iw < Nw4 + 1; iw++)
		{
			data[idx + iw] = data0[idw + iw - Nw1];
		}
	}
}


/*
	����У��
*/
__global__ void PS2d(hipfftComplex* Data, float* Vmin, int iw, int Nx, int Nw, int iz, float w, float dkx, float StepDz)
{
	/*��Kx������������*/
	int ix = threadIdx.x + blockDim.x*blockIdx.x;
	int idw = Nw*ix;
	float kx = 0;
	ix < Nx / 2 ? kx = ix*dkx : kx = -(Nx - ix)*dkx;/*����ת��*/


	hipComplex Factor;
	float ARG = 1.0f - kx*kx*Vmin[iz] * Vmin[iz] / (w*w);/*������������*/
	if (ARG <= 0)
	{
		ARG = 0.0f;
		Factor = make_hipComplex(0.0f, 0.0f);
	}
	else
	{
		Factor = make_hipComplex(cos(w / Vmin[iz] * sqrtf(ARG)*StepDz), sin(w / Vmin[iz] * sqrtf(ARG)*StepDz));
	}

	if (ix < Nx)
	{
		hipComplex d = Data[idw + iw];
		Data[idw + iw] = ComplexMul(d, Factor);
	}

}
__global__ void PS3d(hipfftComplex* Data, float* Vmin, int iw, int Nx, int Ny, int Nw, int iz, float w, float dkx, float dky, float StepDz)
{
	/*��Kx,Ky������������*/
	int ix = threadIdx.x + blockDim.x*blockIdx.x;
	int iy = threadIdx.y + blockDim.y*blockIdx.y;
	int idw = Nw*(iy*Ny + ix);
	float kx = 0;
	float ky = 0;

	iy < Nx / 2 ? kx = iy*dkx : kx = -(Nx - iy)*dkx;/*����ת��*/
	ix < Ny / 2 ? ky = ix*dky : ky = -(Ny - ix)*dky;/*����ת��*/

	hipComplex Factor;
	float ARG = 1.0f - (kx*kx + ky*ky)*Vmin[iz] * Vmin[iz] / (w*w);/*������������*/
	if (ARG <= 0)
	{
		ARG = 0.0f;
		Factor = make_hipComplex(0.0f, 0.0f);
	}
	else
	{
		Factor = make_hipComplex(cos(w / Vmin[iz] * sqrtf(ARG)*StepDz), sin(w / Vmin[iz] * sqrtf(ARG)*StepDz));
	}

	if (ix < Ny&&iy < Nx)
	{
		hipComplex d = Data[idw + iw];
		Data[idw + iw] = ComplexMul(d, Factor);
	}
}
/*
	ʱ��У��
*/
__global__ void SSF2d(hipfftComplex* Data, float* PhaseFactor, float* Vmin, int iw, int Nx, int Nz, int Nw, int iz, float w, float dkx, float StepDz)
{
	/*��Kx,Ky������������*/
	int ix = threadIdx.x + blockDim.x*blockIdx.x;

	int idw = Nw*ix;
	int idz = Nz*ix;

	float kx = 0;

	ix < Nx / 2 ? kx = ix*dkx : kx = -(Nx - ix)*dkx;/*����ת��*/

	if (ix < Nx)
	{
		float ARG = w*PhaseFactor[idz + iz];
		hipfftComplex Factor = make_hipComplex(cos(ARG*StepDz), sin(ARG*StepDz));
		hipComplex d = Data[idw + iw];
		Data[idw + iw] = ComplexMul(d, Factor);
	}
}

__global__ void SSF3d(hipfftComplex* Data, float* PhaseFactor, float* Vmin, int iw, int Nx, int Ny, int Nz, int Nw, int iz, float w, float dkx, float dky, float StepDz)
{
	/*��Kx,Ky������������*/
	int ix = threadIdx.x + blockDim.x*blockIdx.x;
	int iy = threadIdx.y + blockDim.y*blockIdx.y;

	int idw = Nw*(iy*Ny + ix);
	int idz = Nz*(iy*Ny + ix);

	float kx = 0;
	float ky = 0;

	iy < Nx / 2 ? kx = iy*dkx : kx = -(Nx - iy)*dkx;/*����ת��*/
	ix < Ny / 2 ? ky = ix*dky : ky = -(Ny - ix)*dky;/*����ת��*/


	if (ix < Ny&&iy < Nx)
	{
		float ARG = w*PhaseFactor[idz + iz];
		hipfftComplex Factor = make_hipComplex(cos(ARG*StepDz), sin(ARG*StepDz));
		hipComplex d = Data[idw + iw];
		Data[idw + iw] = ComplexMul(d, Factor);
	}
}


/*
��ά����Ҷ���޲������-�ӿں���
*/
void ffdExtrapolation(complex<float>*** Data0, float*** VelModel, float ***PhaseFactor, float* Vmin,
	int Nx, int Ny, int Nz, int Nt,
	int Nw, int  Nw1, int  Nw2, int  Nw3, int  Nw4,
	float dkx, float dky, float dw,
	int StepNum, float StepDz,int NumDepth,
	float** BeamImageXoY,float** BeamImageXoZ,float** BeamImageYoZ,
	int SxGrid,int SyGrid,int wave_len)
{
			
	/*
	����Դ��Ƿ�Խ�磬���ô���
	*/
	size_t freeMem, totalMem; hipMemGetInfo(&freeMem, &totalMem);
	float free_device_memory = freeMem / (1024.0 * 1024.0 * 1024.0);
	float need_device_memory = Nx*Ny*Nt * sizeof(float)+Nx*Ny*(Nt/2+1)*sizeof(hipfftComplex) +Nx*Ny*Nw*sizeof(hipfftComplex)+
			Nx*Ny*Nz * 2 * sizeof(float) + (Nz) * sizeof(float);
	need_device_memory /= (1024.0 * 1024.0 * 1024.0);//GB
	size_t worksize = 0;
	hipfftEstimate3d(Nx, Ny, Nt, HIPFFT_R2C, &worksize); 
	need_device_memory+=worksize/(1024.0 * 1024.0 * 1024.0);
	//��άƽ��FFT
	const int rank = 2;			 
	int n[rank] = { Nx, Ny };
	int inembed[3] = { Nw ,Ny ,Nx }; // �������ݵ�[ҳ��������������](3ά)��[����������]��2ά��
	int onembed[3] = { Nw, Ny,Nx }; // ������ݵ�[ҳ��������������]��[����������]��2ά��
	int istride = Nw; // ÿ�������ź���������Ԫ�صľ���
	int idist = 1;   // ÿ���������źŵ�һ��Ԫ�صľ���
	int ostride = Nw; // ÿ������ź���������Ԫ�صľ���
	int odist = 1;   // ÿ��������źŵ�һ��Ԫ�صľ���
	int batch = Nw;  // ���� fft ���źŸ���
	hipfftEstimateMany(rank,n,inembed,istride,idist, onembed,  ostride,odist, HIPFFT_C2C, batch,&worksize);
	need_device_memory+=worksize/(1024.0 * 1024.0 * 1024.0);

	if (need_device_memory > 15.8)/*�Ƿ񳬹�ȫ���Դ�*/
	{
		printf("Thr program need %fG memory,It's out of device Memory!", need_device_memory);
		cin.get();
		exit(0);
	}
	if ((free_device_memory - need_device_memory) < 0.1)/*�Ƿ񳬹���ǰ�����Դ�,����ֵ0.1G�Դ�*/
	{
		hipDeviceReset();//�����豸���Ա�֤�㹻���Դ���á�
	}
	/*�������豸1ִ����ش���*/
	//hipSetDevice(0);
	//hipDeviceReset();
	/*
	�����ڴ�����
	*/
	float* host_Data0; Check(hipHostMalloc((void**)&host_Data0, Nx*Ny*Nt * sizeof(float)));
	float* host_VelModel;	Check(hipHostMalloc((void**)&host_VelModel, Nx*Ny*Nz * sizeof(float)));
	float* host_PhaseFactor; Check(hipHostMalloc((void**)&host_PhaseFactor, Nx*Ny*Nz * sizeof(float)));
	float* host_Vmin = Vmin;	/*ά����ͬ��ֱ�Ӵ���ָ�뼴��*/
	/*
	��������ά��ת����3D-->1D
	*/
#pragma omp parallel for //����ά�ṹתΪһά�ṹ
	for (int i = 0; i < Nx; i++)
		for (int j = 0; j < Ny; j++)
		{
			for (int k = 0; k < Nt; k++)
			{
				host_Data0[i*(Ny*Nt) + j*Nt + k] = Data0[i][j][k].real();
			}
			for (int k = 0; k < Nz; k++)
			{
				host_VelModel[i*(Ny*Nz) + j*Nz + k] = VelModel[i][j][k];
				host_PhaseFactor[i*(Ny*Nz) + j*Nz + k] = PhaseFactor[i][j][k];
			}
		}

	/*
	�豸�ڴ�����
	*/
	float* device_Data0_real; Check(hipMalloc((void**)&device_Data0_real, Nx*Ny*Nt*sizeof(float)));
	hipfftComplex* device_Data0; Check(hipMalloc((void**)&device_Data0, Nx*Ny*(Nt/2+1)*sizeof(hipfftComplex)));
	hipfftComplex* device_dataW; Check(hipMalloc((void**)&device_dataW, Nx*Ny*Nw * sizeof(hipfftComplex)));/*�洢��Ƶ������*/
	float* device_VelModel; Check(hipMalloc((void**)&device_VelModel, Nx*Ny*Nz * sizeof(float)));
	float* device_PhaseFactor; Check(hipMalloc((void**)&device_PhaseFactor, Nx*Ny*Nz * sizeof(float)));
	float* device_Vmin; Check(hipMalloc((void**)&device_Vmin, Nz * sizeof(float)));

	/*
	��ʼ���豸�ڴ� : Host-->Device
	*/
	printf("Initialization*******\n");
	Check(hipMemcpy(device_Data0_real,host_Data0, Nx*Ny*Nt * sizeof(float),
		hipMemcpyHostToDevice));
	Check(hipMemcpy(device_VelModel, host_VelModel, Nx*Ny*Nz * sizeof(float),
		hipMemcpyHostToDevice));
	Check(hipMemcpy(device_PhaseFactor, host_PhaseFactor, Nx*Ny*Nz * sizeof(float),
		hipMemcpyHostToDevice));
	Check(hipMemcpy(device_Vmin, host_Vmin, Nz * sizeof(float),
		hipMemcpyHostToDevice));

	/*����cufftplan3D��cufftplanmany�����*/
	hipfftHandle cufft3DHandle;
	hipfftResult_t cufftresult;
	cufftresult = hipfftPlan3d(&cufft3DHandle, Nx, Ny, Nt, HIPFFT_R2C);
	/*
	ִ��fft���任  3D (x,y,t)-->(kx,ky,w)
	*/
	hipfftExecR2C(cufft3DHandle, device_Data0_real, device_Data0);
	hipfftDestroy(cufft3DHandle);
	cufftresult = hipfftPlan3d(&cufft3DHandle, Nx, Ny, Nt, HIPFFT_C2R);
	/*
//	����cuFFT���
//	*/
//	hipfftDestroy(cufft3DHandle);

	//��άplan_fft
	hipfftHandle plan_Nxyfft_many;
	cufftresult=hipfftPlanMany(&plan_Nxyfft_many, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch);

	/*
	��ȡ����Ƶ�ʷ�Χ������
	*/
	dim3 g1((Ny + 15) / 16, (Nx + 15) / 16);
	dim3 b1(16, 16);
	HAMMING_Window3d << <g1, b1 >> > (device_Data0, device_dataW, Nw1, Nw2, Nw3, Nw4, Nw,(Nt/2+1), Nx, Ny);

	/*����Ҷ���޲�ֵ���*/
	dim3 g2((Ny + 15) / 16, (Nx + 15) / 16);
	dim3 b2(16, 16);
	for (int iz = 0; iz < StepNum; iz++)
	{
		printf("iz=%d\n", iz);
		/*Step1:����У��,FK��*/
		for (int iw = 0; iw < Nw; iw++)
		{
			float w;
			(iw + Nw1) < Nt / 2 ? w = (iw + Nw1)*dw : w = -(Nt - ((iw + Nw1)))*dw;/*Ƶ��ת��*/
			if (w == 0) w = 0.0000001;
			PS3d << <g2, b2 >> > (device_dataW, device_Vmin, iw, Nx, Ny, Nw, iz, w, dkx, dky, StepDz);
		}

		/*x-o-yƽ�淴FFT�任 (x,y,w)-->(kx,ky,w)*/
		hipfftExecC2C(plan_Nxyfft_many, device_dataW, device_dataW, HIPFFT_BACKWARD); // ִ�� cuFFTplanmany�����任
		dim3 b3 = 512;
		dim3 g3 = ((Nx*Ny*Nw) + b3.x* b3.x - 1) / (b3.x* b3.x);
		normalizing3d << <g3, b3 >> > (device_dataW, Nx*Ny*Nw, Nx*Ny);

		/*Step2:ʱ��У��,FX��*/
		for (int iw = 0; iw < Nw; iw++)
		{
			float w;
			(iw + Nw1) < Nt / 2 ? w = (iw + Nw1)*dw : w = -(Nt - ((iw + Nw1)))*dw;/*Ƶ��ת��*/
			if (w == 0) w = 0.0000001;

			SSF3d << <g2, b2 >> > (device_dataW, device_PhaseFactor, device_Vmin, iw, Nx, Ny, Nz, Nw, iz, w, dkx, dky, StepDz);
		}

		/*x-o-yƽ�淴FFT�任 (kx,ky,w)-->(x,y,w)*/
		hipfftExecC2C(plan_Nxyfft_many, device_dataW, device_dataW, HIPFFT_FORWARD); // ִ�� cuFFTplanmany�����任
		
	//	/*
	//	������Ƶ�ʷ�Χ�����ݷ���
	//	*/
	//	HAMMING_Window_Inverse3d << <g1, b1 >> > (device_Data0, device_dataW, Nw1, Nw2, Nw3, Nw4, Nw,( Nt/2+1), Nx, Ny);
	//	/*
	//	ִ��fft���任  3D (kx,ky,w)-->(x,y,t)
	//	*/
	//	hipfftExecC2R(cufft3DHandle, device_Data0, device_Data0_real);
	//	/*
	//	���ݴ��䣬:Device-->Host
	//	*/
	//	dim3 b4(512);
	//	dim3 g4(((Nx*Ny*Nt) + b4.x* b4.x - 1) / (b4.x* b4.x));
	//	normalizing3d<< <g4, b4 >> > (device_Data0_real, Nx*Ny*Nt, Nx*Ny*Nt);
	//	Check(hipMemcpy(host_Data0, device_Data0_real, Nx*Ny*Nt * sizeof(float), hipMemcpyDeviceToHost));

	//	/*
	//	��������ά��ת����1D-->3D
	//	*/
	//	//#pragma omp parallel for //����ά�ṹתΪһά�ṹ
	//	for (int i = 0; i < Nx; i++)
	//		for (int j = 0; j < Ny; j++)
	//			for (int k = 0; k < Nt; k++)
	//			{
	//				Data0[i][j][k] = (host_Data0[i*(Ny*Nt) + j*Nt + k], 0);
	//			}


	//	//����XoY���
	//	if (iz ==  NumDepth)
	//	{
	//		for (int i = 0; i < Nx; i++)
	//			for (int j = 0; j < Ny; j++)
	//			{
	//				BeamImageXoY[i][j] = Data0[i][j][wave_len].real();
	//			}
	//	}
	//	
	//	//����XoZ���
	//	for (int i = 0; i < Nx; i++)
	//	{
	//		BeamImageXoZ[i][iz] = Data0[i][SyGrid][wave_len].real();
	//	}
	//	//����YoZ���
	//	for (int j = 0; j <Ny; j++)
	//	{
	//		BeamImageYoZ[j][iz] = Data0[SxGrid][j][wave_len].real();
	//	}
	}

	/*
	�ͷ�����/�豸�ڴ�
	*/
	if (host_Data0 != 0)			hipHostFree(host_Data0);
	if (host_VelModel != 0)			hipHostFree(host_VelModel);
	if (host_PhaseFactor != 0)		hipHostFree(host_PhaseFactor);

	if (device_Data0 != 0)			hipFree(device_Data0);
	if (device_dataW != 0)			hipFree(device_dataW);
	if (device_VelModel != 0)		hipFree(device_VelModel);
	if (device_PhaseFactor != 0)	hipFree(device_PhaseFactor);
	if (device_Vmin != 0)			hipFree(device_Vmin);
	
	hipfftDestroy(cufft3DHandle);
	hipfftDestroy(plan_Nxyfft_many);
}
